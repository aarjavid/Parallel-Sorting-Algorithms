#include<stdio.h>
#include<hip/hip_runtime.h>
#include <time.h>

typedef  unsigned int    idata;

void printData (idata *d,int N);
void readDataFromFile(idata *d,int N,char *);

void readDataFromFile(idata *d,int N,char *myfilename) 
{
    //printf("Reading Data From File\n");
    FILE* fp = fopen (myfilename, "r");
    unsigned int val = 0;
    int i;
    if (fp == NULL)
      {
         puts("Couldn't open file");
         exit(0);
      }
    else
    {   
        for (i=0;i<N;i++)
        {  
            fscanf (fp, "%d", &val);   
            d[i] = val;
            //printf ("%d\n", val);     
        }
       // printf("Reading Input Over\n");
        fclose (fp);    
    } 
}

void printData (idata *d,int N) 
{
    int i;
    for (i=0;i<N;i++) {
        printf("%d  ",d[i]);
    }
    printf("\n");

}

void printElaspedTime(clock_t start,clock_t end) 
{
    double cpu_time_used;
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    //printf("\n\nRunning Time=%f seconds\n\n",cpu_time_used);
    printf("\nsort,oets,cuda,%f\n",cpu_time_used);

}


__global__ void even_kernel(idata* d,int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	index = index * 2 ;
    int temp;
    
	if(index <=  N-2){
		if(d[index]>d[index+1]){
			temp=d[index];
			d[index]=d[index+1];
			d[index+1]=temp;
		}
	}
	
}

__global__ void odd_kernel(idata* d,int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	index = index * 2 + 1;
    int temp;
    
	if(index <=  N-2){
		if(d[index]>d[index+1]){
			temp=d[index];
			d[index]=d[index+1];
			d[index+1]=temp;
		}
	}
	
}

void oets(idata *d,int N) 
{
    clock_t start, end;
	idata *cuda_inpoup;
	int i;
	hipMalloc((void**)&cuda_inpoup, N*sizeof(idata));
	hipMemcpy(cuda_inpoup,d,N*sizeof(idata),hipMemcpyHostToDevice);
    start = clock();

	int num_of_blocks =int( ceil((double)N/1024));
	int num_of_threads = (N<1024?N:1024);
	for(i=0;i<=N/2;i++){
		even_kernel<<<num_of_blocks,num_of_threads>>>(cuda_inpoup,N);
    	odd_kernel<<<num_of_blocks,num_of_threads>>>(cuda_inpoup,N);
	}
    hipDeviceSynchronize();
    end = clock();
	hipMemcpy(d,cuda_inpoup,N*sizeof(idata), hipMemcpyDeviceToHost);
	hipFree(d);
    printElaspedTime(start,end);

}

int main(int argc, char *argv[]) {
  
    int N;
    idata  *sptr;
    sptr = NULL;
    char *filename = NULL;
	
    N   = atoi(argv[1]);
    filename = argv[2];

    fprintf(stdout,"N=%d filename=%s\n",N,filename);
    sptr  = (idata *) malloc(N*sizeof(idata));
    
    readDataFromFile(sptr,N,filename);
    //printf("Input Data is:-\n");
    //printData(sptr,N);

    oets(sptr,N);
    //printf("Sorted Array is:\n");
    //printData(sptr,N);
	return 0;

}

