#include<stdio.h>
#include<hip/hip_runtime.h>
#include <time.h>

typedef  unsigned int    idata;

void printData (idata *d,int N);
void readDataFromFile(idata *d,int N,char *);

void readDataFromFile(idata *d,int N,char *myfilename) 
{
    //printf("Reading Data From File\n");
    FILE* fp = fopen (myfilename, "r");
    unsigned int val = 0;
    int i;
    if (fp == NULL)
      {
         puts("Couldn't open file");
         exit(0);
      }
    else
    {   
        for (i=0;i<N;i++)
        {  
            fscanf (fp, "%d", &val);   
            d[i] = val;
            //printf ("%d\n", val);     
        }
        //printf("Reading Input Over\n");
        fclose (fp);    
    } 
}

void printData (idata *d,int N) 
{
    int i;
    for (i=0;i<N;i++) {
        printf("%d  ",d[i]);
    }
    printf("\n");

}

void printElaspedTime(clock_t start,clock_t end) 
{
    double cpu_time_used;
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    //printf("\n\nRunning Time=%f seconds\n\n",cpu_time_used);
    printf("\nsort,btns,cuda,%f\n",cpu_time_used);

}

__global__ void bitonic_kernel(idata* d,int cd,int cbs)
{
    idata temp;
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i^cd;
    if(j>i)
    {
        if((i&cbs)==0) //Sort in Increasing order
        {
            if(d[i]>d[j])
            {
                temp = d[i];
                d[i] = d[j];
                d[j] = temp;
                
            }
        }
        else //Sort in decreasing order
        {
            if(d[i]<d[j])
            {
                temp = d[i];
                d[i] = d[j];
                d[j] = temp;
                
            }
        }
    }
}


void btns(idata *d,int N) 
{
    clock_t start, end;
	idata *cuda_inpoup;
	hipMalloc((void**)&cuda_inpoup, N*sizeof(idata));
	hipMemcpy(cuda_inpoup,d,N*sizeof(idata),hipMemcpyHostToDevice);
    start = clock();
	int num_of_blocks =int( ceil((double)N/1024));
	int num_of_threads = N/num_of_blocks;

    int cbs; //current bitonic sequence length
    int cd; //comparison distance
    
    for(cbs=2; cbs<=N; cbs=cbs<<1) 
    {
        for(cd=cbs>>1; cd>0; cd=cd>>1)
        {
            bitonic_kernel<<<num_of_blocks,num_of_threads>>>(cuda_inpoup,cd,cbs);
        }
    }
    hipDeviceSynchronize();
    end = clock();
	hipMemcpy(d,cuda_inpoup,N*sizeof(idata), hipMemcpyDeviceToHost);
	hipFree(d);
    printElaspedTime(start,end);
}


int main(int argc, char *argv[]) {
  
    int N;
    idata  *sptr;
    sptr = NULL;
    char *filename = NULL;
	//clock_t start, end;
	
    N   = atoi(argv[1]);
    filename = argv[2];

    fprintf(stdout,"N=%d filename=%s\n",N,filename);
    sptr  = (idata *) malloc(N*sizeof(idata));
    
    readDataFromFile(sptr,N,filename);
    //printf("Input Data is:-\n");
    //printData(sptr,N);

    btns(sptr,N);
    //printf("Sorted Array is:\n");
    //printData(sptr,N);
	return 0;

}

