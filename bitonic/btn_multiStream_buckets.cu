#include<stdio.h>
#include<hip/hip_runtime.h>
#include <time.h>

typedef  unsigned int    idata;

void printData (idata *d,int N);
void readDataFromFile(idata *d,int N,char *);

void readDataFromFile(idata *d,int N,char *myfilename) 
{
    //printf("Reading Data From File\n");
    FILE* fp = fopen (myfilename, "r");
    unsigned int val = 0;
    int i;
    if (fp == NULL)
      {
         puts("Couldn't open file");
         exit(0);
      }
    else
    {   
        for (i=0;i<N;i++)
        {  
            fscanf (fp, "%d", &val);   
            d[i] = val;
            //printf ("%d\n", val);     
        }
       // printf("Reading Input Over\n");
        fclose (fp);    
    } 
}

void printData (idata *d,int N) 
{
    int i;
    for (i=0;i<N;i++) {
        printf("%d  ",d[i]);
    }
    printf("\n");

}

void printElaspedTime(clock_t start,clock_t end) 
{
    double cpu_time_used;
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    //printf("\n\nRunning Time=%f seconds\n\n",cpu_time_used);
    printf("\nsort,oets,cuda,%f\n",cpu_time_used);

}

__global__ void bitonic_kernel(idata* d,int cd,int cbs)
{
    idata temp;
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i^cd;
    if(j>i)
    {
        if((i&cbs)==0) //Sort in Increasing order
        {
            if(d[i]>d[j])
            {
                temp = d[i];
                d[i] = d[j];
                d[j] = temp;
                
            }
        }
        else //Sort in decreasing order
        {
            if(d[i]<d[j])
            {
                temp = d[i];
                d[i] = d[j];
                d[j] = temp;
                
            }
        }
    }
}


__global__ void null_kernel() {
	//do nothing
}

void btns(idata *cuda_inpoup,int N,int num_streams,int cbsi) 
{

	int num_of_blocks =int( ceil((double)N/1024));
	int num_of_threads = N/num_of_blocks;

	int cbs; //current bitonic sequence length
	int cd; //comparison distance
	
	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) {
		hipStreamCreate(&streams[i]);
    		for(cbs=cbsi; cbs<=N; cbs=cbs<<1) 
    		{
        		for(cd=cbs>>1; cd>0; cd=cd>>1)
        		{
           		 bitonic_kernel<<<num_of_blocks,num_of_threads,0,streams[i]>>>(cuda_inpoup,cd,cbs);
        		}
    		}

	}
}
__global__ void even_kernel(idata* d,int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	index = index * 2 ;
    int temp;
    
	if(index <=  N-2){
		if(d[index]>d[index+1]){
			temp=d[index];
			d[index]=d[index+1];
			d[index+1]=temp;
		}
	}
	
}

__global__ void odd_kernel(idata* d,int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	index = index * 2 + 1;
    int temp;
    
	if(index <=  N-2){
		if(d[index]>d[index+1]){
			temp=d[index];
			d[index]=d[index+1];
			d[index+1]=temp;
		}
	}
	
}

void oets(idata *cuda_inpoup,int Ns, int num_streams) 
{

	int num_of_blocks =int( ceil((double)Ns/1024));
	int num_of_threads = (Ns<1024?Ns:1024);
	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) {
		hipStreamCreate(&streams[i]);
 
		for(int j=0;j<=Ns/2;j++){
			even_kernel<<<num_of_blocks,num_of_threads,0,streams[i] >>>(cuda_inpoup+i*Ns,Ns);
    			odd_kernel<<<num_of_blocks,num_of_threads,0,streams[i] >>>(cuda_inpoup+i*Ns,Ns);
			}
		
	}

}

__global__ void swap(idata* d,int N) {

	int j = blockDim.x * blockIdx.x + threadIdx.x ;
	idata temp;
	if(j <  N/2){
		temp = d[j];
		d[j] = d[N-1-j];
		d[N-1-j] = temp;
	}
}

void reverse(idata *d,int Ns) {
	int num_of_blocks =int( ceil((double)Ns/1024));
	int num_of_threads = (Ns<1024?Ns:1024);
	swap<<<num_of_blocks,num_of_threads>>>(d,Ns);
}

int main(int argc, char *argv[]) {
  
    int N;
    idata  *sptr,*aptr;
    sptr = NULL;
    aptr = NULL;
    char *filename = NULL;
	clock_t start, end;
	
    N   = atoi(argv[1]);
    filename = argv[2];
    int num_batches   = atoi(argv[3]);

    fprintf(stdout,"N=%d filename=%s\n",N,filename);
    sptr  = (idata *) malloc(N*sizeof(idata));
    
    readDataFromFile(sptr,N,filename);
    //printf("Input Data is:-\n");
    //printData(sptr,N);


    idata *cuda_inpoup;
    hipMalloc(&cuda_inpoup, N * sizeof(idata));
    hipMemcpy(cuda_inpoup,sptr,N*sizeof(idata), hipMemcpyHostToDevice);

    start = clock();
    int num_streams = num_batches;
    int N_per_stream = N/num_streams;
    aptr = cuda_inpoup;
    int cbs=2;
    btns(aptr,N_per_stream,num_streams,cbs);


    hipDeviceSynchronize(); 
    
    num_streams = 1; 
    N_per_stream = 2*N/num_batches; 
    int N_rev=N_per_stream/2; 
    cbs = N_per_stream;
    for(int i=0;i<num_batches/2;i++) { 
			              
       for(int j=0;j<num_batches/2;j++) {   //even round 
       		aptr = cuda_inpoup + (2*j*N)/num_batches; 
 	        reverse(aptr+N_rev,N_rev); 
 		btns(aptr,N_per_stream,num_streams,cbs); 
   	} 
        
	hipDeviceSynchronize(); 
  	       
  	for(int j=0;j+1<num_batches/2;j++) { // odd round 
  		 aptr = cuda_inpoup + (((2*j)+1)*N)/num_batches; 
                 reverse(aptr+N_rev,N_rev); 
                 btns(aptr,N_per_stream,num_streams,cbs); 
   	} 
   			        
        hipDeviceSynchronize(); 
    } 
    
    end = clock();
    hipMemcpy(sptr,cuda_inpoup,N*sizeof(idata), hipMemcpyDeviceToHost);
//    printf("Sorted Array is:\n");
//    printData(sptr,N);
    printElaspedTime(start,end);
	return 0;

}

